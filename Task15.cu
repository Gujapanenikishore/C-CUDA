

/*Task 15: Statistical Computation with Modern C++ (Individual)

Implement mean, median, variance, and standard deviation calculations using modern C++17 features.

Clearly illustrate computations and validate with known data sets.*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <numeric>
#include <cmath>
#include <iomanip>  // for std::setprecision

// ---------- Utility Functions -------------
double computeMean(const std::vector<double>& data) {
    return std::accumulate(data.begin(), data.end(), 0.0) / data.size();
}

double computeMedian(std::vector<double> data) {
    std::sort(data.begin(), data.end());
    size_t n = data.size();
    return (n % 2 == 0) ? (data[n / 2 - 1] + data[n / 2]) / 2.0 : data[n / 2];
}

double computeVariance(const std::vector<double>& data, bool sample = false) {
    double mean = computeMean(data);
    double sum_sq = 0.0;
    for (double x : data)
        sum_sq += (x - mean) * (x - mean);
    return sum_sq / (data.size() - (sample ? 1 : 0));
}

double computeStdDev(const std::vector<double>& data, bool sample = false) {
    return std::sqrt(computeVariance(data, sample));
}

// --------- Display Results ---------------
void displayStats(const std::vector<double>& data) {
    std::cout << "Data: ";
    for (double x : data) std::cout << x << " ";
    std::cout << "\nSize: " << data.size() << '\n';

    double mean = computeMean(data);
    double median = computeMedian(data);
    double variance = computeVariance(data);
    double std_dev = computeStdDev(data);

    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Mean              : " << mean << "\n";
    std::cout << "Median            : " << median << "\n";
    std::cout << "Variance (Pop)    : " << variance << "\n";
    std::cout << "Std Dev (Pop)     : " << std_dev << "\n";
    std::cout << "Variance (Sample) : " << computeVariance(data, true) << "\n";
    std::cout << "Std Dev (Sample)  : " << computeStdDev(data, true) << "\n";
}

int main() {
    // === Example dataset ===
    std::vector<double> dataset1 = {10, 20, 30, 40, 50};      // known data
    std::vector<double> dataset2 = {2.5, 3.1, 4.0, 3.9, 2.7};  // real numbers

    std::cout << "=== Dataset 1 ===\n";
    displayStats(dataset1);

    std::cout << "\n=== Dataset 2 ===\n";
    displayStats(dataset2);

    return 0;
}

/*_________________
OUTPUT

=== Dataset 1 ===
Data: 10 20 30 40 50 
Size: 5
Mean              : 30.0000
Median            : 30.0000
Variance (Pop)    : 200.0000
Std Dev (Pop)     : 14.1421
Variance (Sample) : 250.0000
Std Dev (Sample)  : 15.8114

=== Dataset 2 ===
Data: 2.5 3.1 4 3.9 2.7 
Size: 5
Mean              : 3.2400
Median            : 3.1000
Variance (Pop)    : 0.3484
Std Dev (Pop)     : 0.5903
Variance (Sample) : 0.4355
Std Dev (Sample)  : 0.6600*/